#include "hip/hip_runtime.h"


#include "cuda_macros.h"
#include "util.h"

namespace pocs_cuda {

void
ifft_stack (float *S, int M, int N, int C)
{
	hipfftHandle plan;
	int c;

	cufft (Plan2d (&plan, N, M, HIPFFT_C2C));

	for (c=0; c<C; c++){
		cufft (ExecC2C
			(plan,
			 (hipfftComplex*)(S + c*2*M*N),
			 (hipfftComplex*)(S + c*2*M*N),
			 HIPFFT_BACKWARD));
	}

	scale (S, C*M*N, 1.0f/sqrtf(M*N), 0);

	cufft (Destroy (plan));
}

};
